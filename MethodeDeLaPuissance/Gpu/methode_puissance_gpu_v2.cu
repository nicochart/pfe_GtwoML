#include "hip/hip_runtime.h"
/* Méthode de la puissance GPU */
/* Auteur: C. Bouillaguet et P. Fortin (Univ. Lille) code séquentiel | D.Leroye & N.Hochart code GPU */
/* Compilation : nvcc nom.cu -o nom --generate-code arch=compute_61,code=sm_61 -O3 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#include "defs.h"

//Taille des blocs : donne des blocs de 256 threads
#define TAILLE_BLOC_X 1
#define TAILLE_BLOC_Y 256

// Initialise la ligne N°i, à n éléments, de la matrice A :
__host__ __device__ void init_ligne_gpu(REAL_T *A, long i, long n)
{
  for (long j = 0; j < n; j++)
	{
    A[i*n+j] = (((REAL_T)((i * i * PRNG_1 + j * j * PRNG_2) & PRNG_MAX)) / PRNG_MAX) / n;
  }
  for (long k = 1; k < n; k *= 2)
	{
    if (i + k < n)
		{
      A[i*n + i+k] = ((i - k) * PRNG_2 + i * PRNG_1) % RANGE;
    }
    if (i - k >= 0)
		{
      A[i*n +i-k] = ((i + k) * PRNG_2 + i * PRNG_1) % RANGE;
    }
  }
}

/*kernels*/

//Kernel 1 : calcule le produit matrice vecteur
__global__ void prodmatvectKernel(REAL_T *d_A, REAL_T *d_X, REAL_T *d_Y, int n)
{
	unsigned int i = blockDim.x*blockIdx.x+threadIdx.x;
  long j;

	if (i < n)
	{
		REAL_T temp=0;
		for(j=0;j<n;j++)
		{
			temp += d_A[i*n+j] * d_X[j];
		}
		d_Y[i] = temp;
    }
}

//Kernel 2 : Somme des éléments d'un vecteur pour calcul de norme par réduction. k permet de savoir à quelle étape on est.
__global__ void somme_elements_vecteur(REAL_T *d_vect, int k, int n, REAL_T *d_norm)
{
		unsigned int i = blockDim.x*blockIdx.x+threadIdx.x;
		if (i<n)
		{
			if (i % k == 0)
			{
				d_vect[i] = fabs(d_vect[i]) + fabs(d_vect[i + k/2]);
			}
		}
    *d_norm = d_vect[0]; //à chaque fois, on copie la valeur à l'indice 0 dans un autre vecteur. Si on est pas à la dernière étape de la reduction, cette valeur n'a pas de sens.
}

//Kernel 3 : Divise l'ensemble des éléments d'un vecteur par un reel et calcule le vecteur erreur contenant les erreurs locales
__global__ void normAndError(REAL_T *d_Y, REAL_T *d_normY,  REAL_T *d_X, REAL_T *d_Err, int n)
{
	unsigned int i = blockDim.x*blockIdx.x+threadIdx.x;
	if (i < n)
	{
		d_Y[i] = d_Y[i] / *d_normY;
    d_Err[i] = fabs(d_X[i] - d_Y[i]);
  }
}

//Kernel 4 : Génération de morceaux de matrice directement sur GPU
__global__ void matrixInit(REAL_T *d_A, int n)
{
	unsigned int i = blockDim.x*blockIdx.x+threadIdx.x;
	if (i < n)
	{
		init_ligne_gpu(d_A, i, n); //initialisation de la ligne i de d_A (gpu)
  }
}

//Kernel 5 : Initialisation du vecteur X à 1/n sur GPU
__global__ void vectorInit(REAL_T *d_X, int n)
{
	unsigned int i = blockDim.x*blockIdx.x+threadIdx.x;
	if (i < n)
	{
		d_X[i] = 1.0 / (double) n;
	}
}

//Kernel 6 : Copie de vect2 dans vect1
__global__ void vectorCopy(REAL_T *d_vect1, REAL_T *d_vect2, int n)
{
	unsigned int i = blockDim.x*blockIdx.x+threadIdx.x;
	if (i < n)
	{
		d_vect1[i] = d_vect2[i];
	}
}

int main(int argc, char **argv)
{
    int i,k,n;
    long long size, size_vector;
    REAL_T norm, error;
    REAL_T *d_error, *d_norm, *tmp, *X, *d_A, *d_X, *d_Y, *d_Y_tmpsum, *d_Err;
    double start_time, total_time;
    int n_iterations;
    FILE *output;

    if (argc < 2)
		{
        printf("USAGE: %s [n]\n", argv[0]);
        exit(1);
    }
    n = atoi(argv[1]);
    size = (long long) n * n * sizeof(REAL_T);
    size_vector = (long long) n * sizeof(REAL_T);
    printf("taille de la matrice : %.1f G\n", size / 1073741824.);

    // Allocation CPU
    X = (REAL_T *)malloc(n * sizeof(REAL_T));
    if (X == NULL)
    {
        perror("impossible d'allouer le vecteur X");
        exit(1);
    }

    // Allocation GPU
    hipMalloc((void **) &d_A, size);
    hipMalloc((void **) &d_X, size_vector);
    hipMalloc((void **) &d_Y, size_vector);
		hipMalloc((void **) &d_Y_tmpsum, size_vector);
    hipMalloc((void **) &d_Err, size_vector);
    hipMalloc((void **) &d_norm, sizeof(REAL_T));
    hipMalloc((void **) &d_error, sizeof(REAL_T));

    //Initialisation des kernels
    dim3 threadsParBloc(TAILLE_BLOC_Y);
    dim3 tailleGrille(TAILLE_BLOC_Y);

		/*** initialisation de la matrice ***/
		//Lancement de Kernel 4 : génération de la matrice
		matrixInit<<<tailleGrille,threadsParBloc>>>(d_A, n);
		/*** initialisation de x ***/
		//Lancement de Kernel 5 : initialisation de X
		vectorInit<<<tailleGrille,threadsParBloc>>>(d_X, n);

    start_time = my_gettimeofday();
    error = 10;
    n_iterations = 0;
    while(error > ERROR_THRESHOLD)
    {
        printf("iteration %4d, erreur actuelle %g\n", n_iterations, error);

        /*** y <--- A.x ***/
        //Lancement de Kernel 1 (asynchrone) (calcul vecteur y)
        prodmatvectKernel<<<tailleGrille,threadsParBloc>>>(d_A,d_X,d_Y,n);

				//Lancement de Kernel 6 (copie de d_Y dans d_Y_tmpsum)
				vectorCopy<<<tailleGrille,threadsParBloc>>>(d_Y_tmpsum, d_Y, n);

				for (k=2;k<=n;k*=2)
				{
        	//Plusieurs lancements du Kernel 2 (étapes de réduction pour calcul de norme)
        	somme_elements_vecteur<<<tailleGrille,threadsParBloc>>>(d_Y_tmpsum,2,n,d_norm);
				}

        /*** y <--- y / ||y||   &   vecteur erreur ***/
        //Lancement de Kernel 3 (normalisation + calul du vecteur erreur)
        normAndError<<<tailleGrille,threadsParBloc>>>(d_Y,d_norm,d_X,d_Err,n);

        /*** error <--- ||x - y|| ***/
				for (k=2;k<=n;k*=2)
				{
					//Plusieurs lancements du Kernel 2 (étapes de réduction pour calcul de norme)
        	somme_elements_vecteur<<<tailleGrille,threadsParBloc>>>(d_Err,k,n,d_error);
				}

        //transfert GPU -> CPU de d_error vers error
        hipMemcpy(&error, d_error, sizeof(REAL_T), hipMemcpyDeviceToHost);

        /*** x <--> y ***/
        tmp = d_X; d_X = d_Y ; d_Y = tmp;

        n_iterations++;
    }

    //transfert GPU -> CPU de d_X vers X et d_norm vers norm
    hipMemcpy(X, d_X, size_vector, hipMemcpyDeviceToHost);
    hipMemcpy(&norm, d_norm, sizeof(REAL_T), hipMemcpyDeviceToHost);

    total_time = my_gettimeofday() - start_time;
    printf("erreur finale après %4d iterations : %g (|VP| = %g)\n", n_iterations, error, norm);
    printf("temps : %.1f s      Mflop/s : %.1f \n", total_time, (2.0 * n * n + 7.0 * n) * n_iterations / 1048576. / total_time);

    /*** stocke le vecteur propre dans un fichier ***/
    output = fopen("result.out", "w");
    if (output == NULL)
    {
        perror("impossible d'ouvrir result.out en écriture");
        exit(1);
    }
    fprintf(output, "%d\n", n);
    for (i = 0; i < n; i++)
    {
        fprintf(output, "%.17g\n", X[i]);
    }
    fclose(output);

    hipFree(d_A);
    hipFree(d_X);
    hipFree(d_Y);
		hipFree(d_Y_tmpsum);
    hipFree(d_Err);
    hipFree(d_norm);
    hipFree(d_error);
    free(X);
}
