#include "hip/hip_runtime.h"
/* Méthode de la puissance GPU */
/* Auteur: C. Bouillaguet et P. Fortin (Univ. Lille) code séquentiel | D.Leroye & N.Hochart code GPU */
/* Compilation : nvcc nom.cu -o nom --generate-code arch=compute_61,code=sm_61 -O3 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#include "defs.h"

//Taille des blocs : donne des blocs de 256 threads
#define TAILLE_BLOC_X 1
#define TAILLE_BLOC_Y 256

/*kernels*/

//Kernel 1 : calcule le produit matrice vecteur
__global__ void prodmatvectKernel(REAL_T *d_A, REAL_T *d_X, REAL_T *d_Y, int n)
{
	unsigned int i = blockDim.x*blockIdx.x+threadIdx.x;
    long j;

	if (i < n)
	{
		REAL_T temp=0;
		for(j=0;j<n;j++)
		{
			temp += d_A[i*n+j] * d_X[j];
		}
		d_Y[i] = temp;
    }
}


//Kernel 2 : calcule la somme des éléments d'un vecteur
__global__ void somme_elements_vecteur(REAL_T *d_Y, int n, REAL_T *d_somme)
{
    long i;

    REAL_T somme_temp = 0;
    for (i=0; i<n;i++)
    {
        somme_temp += fabs(d_Y[i]);
    }
    *d_somme = somme_temp;
}

//Kernel 3 : Divise l'ensemble des éléments d'un vecteur par un reel et calcule le vecteur erreur contenant les erreurs locales
__global__ void normAndError(REAL_T *d_Y, REAL_T *d_normY,  REAL_T *d_X, REAL_T *d_Err, int n)
{
	unsigned int i = blockDim.x*blockIdx.x+threadIdx.x;
	if (i < n)
	{
		d_Y[i] = d_Y[i] / *d_normY;
        d_Err[i] = fabs(d_X[i] - d_Y[i]);
    }
}

int main(int argc, char **argv)
{
    int i, n;
    long long size, size_vector;
    REAL_T norm, error;
    REAL_T *d_error, *d_norm, *tmp, *A, *X, *d_A, *d_X, *d_Y, *d_Err;
    double start_time, total_time;
    int n_iterations;
    FILE *output;

    if (argc < 2) {
        printf("USAGE: %s [n]\n", argv[0]);
        exit(1);
    }
    n = atoi(argv[1]);
    size = (long long) n * n * sizeof(REAL_T);
    size_vector = (long long) n * sizeof(REAL_T);
    printf("taille de la matrice : %.1f G\n", size / 1073741824.);

    // Allocation CPU
    /*** allocation de la matrice et des vecteurs ***/
    A = (REAL_T *)malloc(size);
    if (A == NULL) {
        perror("impossible d'allouer la matrice");
        exit(1);
    }

    X = (REAL_T *)malloc(n * sizeof(REAL_T));
    if (X == NULL)
    {
        perror("impossible d'allouer le vecteur X");
        exit(1);
    }

    /*** initialisation de la matrice et de x ***/
    for (i = 0; i < n; i++)
    {
        init_ligne(A, i, n);
    }

    for (i = 0; i < n; i++)
    {
        X[i] = 1.0 / n;
    }

    // Allocation GPU
    hipMalloc((void **) &d_A, size);
    hipMalloc((void **) &d_X, size_vector);
    hipMalloc((void **) &d_Y, size_vector);
    hipMalloc((void **) &d_Err, size_vector);
    hipMalloc((void **) &d_norm, sizeof(REAL_T));
    hipMalloc((void **) &d_error, sizeof(REAL_T));

    //Transfert CPU -> GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_X, X, size_vector, hipMemcpyHostToDevice);

    //Initialisation des kernels
    dim3 threadsParBloc(TAILLE_BLOC_Y);
    dim3 tailleGrille(TAILLE_BLOC_Y);

    start_time = my_gettimeofday();
    error = 10;
    n_iterations = 0;
    while(error > ERROR_THRESHOLD)
    {
        printf("iteration %4d, erreur actuelle %g\n", n_iterations, error);

        /*** y <--- A.x ***/
        //Lancement de Kernel 1 (asynchrone) (calcul vecteur y)
        prodmatvectKernel<<<tailleGrille,threadsParBloc>>>(d_A,d_X,d_Y,n);

        //Lancement de Kernel 2 (calcul de la norme de Y)
        somme_elements_vecteur<<<1,1>>>(d_Y,n,d_norm);

        /*** y <--- y / ||y||   &   vecteur erreur***/
        //Lancement de Kernel 3 (normalisation + calul du vecteur erreur)
        normAndError<<<tailleGrille,threadsParBloc>>>(d_Y,d_norm,d_X,d_Err,n);

        /*** error <--- ||x - y|| ***/
        //Lancement de Kernel 4 (calcul de la norme de d_Err)
        somme_elements_vecteur<<<1,1>>>(d_Err,n,d_error);
        //transfert GPU -> CPU de d_error vers error
        hipMemcpy(&error, d_error, sizeof(REAL_T), hipMemcpyDeviceToHost);

        /*** x <--> y ***/
        tmp = d_X; d_X = d_Y ; d_Y = tmp;

        n_iterations++;
    }

    //transfert GPU -> CPU de d_X vers X et d_norm vers norm
    hipMemcpy(X, d_X, size_vector, hipMemcpyDeviceToHost);
    hipMemcpy(&norm, d_norm, sizeof(REAL_T), hipMemcpyDeviceToHost);

    total_time = my_gettimeofday() - start_time;
    printf("erreur finale après %4d iterations : %g (|VP| = %g)\n", n_iterations, error, norm);
    printf("temps : %.1f s      Mflop/s : %.1f \n", total_time, (2.0 * n * n + 7.0 * n) * n_iterations / 1048576. / total_time);

    /*** stocke le vecteur propre dans un fichier ***/
    output = fopen("result.out", "w");
    if (output == NULL)
    {
        perror("impossible d'ouvrir result.out en écriture");
        exit(1);
    }
    fprintf(output, "%d\n", n);
    for (i = 0; i < n; i++)
    {
        fprintf(output, "%.17g\n", X[i]);
    }
    fclose(output);

    hipFree(d_A);
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_Err);
    hipFree(d_norm);
    hipFree(d_error);
    free(A);
    free(X);
}
